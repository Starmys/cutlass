#include <iostream>
#include <fstream>
#include <sstream>
#include <filesystem>
#include <vector>
#include <unordered_map>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemv.h"
#include "cutlass/gemm/kernel/gemv.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "pit_gemv_device.h"
#include "pit_gemv_kernel.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

template<typename ElementIn, typename ElementOut>
int load_array_from_file(ElementOut* arr, std::string filepath) {
  std::cout << "Loading: " << filepath << std::endl;
  std::ifstream infile(filepath);
  int length = 0;
  ElementIn current_number;
  while (infile >> current_number) {
    arr[length++] = current_number;
  }
  return length;
}

/// Result structure
struct Result {

  double diff;
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  Result(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  int batch, K, N, nnz, rows;
  int iterations;

  //
  // Methods
  // 

  Options():
    help(false),
    batch(1),
    K(13824),
    N(5120),
    rows(5000),
    iterations(20)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("batch", batch, 1);
    cmd.get_cmd_line_argument("K", K, 13824);
    cmd.get_cmd_line_argument("N", N, 5120);
    cmd.get_cmd_line_argument("rows", rows, 5000);
    cmd.get_cmd_line_argument("iterations", iterations, 20);
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of real-valued multiply-adds 
    int64_t fmas = (int64_t)batch * (int64_t)K * (int64_t)N;

    // Two flops per multiply-add
    // TODO: calc by thread block
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "97_pit_gemv\n\n"
      << "  This example profiles the performance of a PIT GEMV kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --batch=<int>               Sets the batch size.\n"
      << "  --N=<int>                   Sets the N dimension.\n"
      << "  --K=<int>                   Sets the K dimension.\n"
      << "  --rows=<int>                Sets the number of selected rows.\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n";

    out << "\n\nExamples:\n\n"

      << "# Runs a 13824x5120 PIT GEMV\n"
      << "$ ./examples/97_pit_gemv/97_pit_gemv --batch=1 --N=5120 --K=13824 --rows=5000\n\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Gemv>
class Testbed {
public:

  //
  // Type definitions
  //

  using ElementA = typename Gemv::ElementA;
  using ElementB = typename Gemv::ElementB;
  using ElementC = typename Gemv::ElementC;
  using ElementAccumulator = typename Gemv::ElementAccumulator;

  using LayoutA = typename Gemv::LayoutA;
  using LayoutB = cutlass::layout::RowMajor;
  using LayoutC = cutlass::layout::RowMajor;

  using MatrixCoord = typename LayoutC::TensorCoord;

private:

  //
  // Data members
  //

  Options options;

  cutlass::HostTensor<ElementA, LayoutA> tensor_weight;
  cutlass::HostTensor<ElementB, LayoutB> tensor_activations;
  cutlass::HostTensor<int16_t, LayoutB> tensor_rows;
  cutlass::HostTensor<ElementC, LayoutC> tensor_bias;
  cutlass::HostTensor<ElementC, LayoutC> tensor_output;

  cutlass::HostTensor<ElementC, LayoutC> reference_output;

public:

  Testbed(Options const &options_):
    options(options_) { }

private:

  /// Initializes data structures
  void initialize_() {
    tensor_weight.resize({options.K, options.N});
    tensor_activations.resize({options.batch, options.K});
    tensor_rows.resize({options.batch, options.K});
    tensor_bias.resize({options.batch, options.N});
    tensor_output.resize({options.batch, options.N});

    reference_output.resize({options.batch, options.N});

    std::string data_folder = "examples/97_pit_gemv/data/";
    load_array_from_file<float, ElementA>(tensor_weight.host_ref().data(), data_folder + "weight.txt");
    load_array_from_file<float, ElementB>(tensor_activations.host_ref().data(), data_folder + "activations.txt");
    load_array_from_file<int, int16_t>(tensor_rows.host_ref().data(), data_folder + "rows.txt");
    load_array_from_file<float, ElementC>(reference_output.host_ref().data(), data_folder + "output.txt");
    cutlass::reference::host::TensorFill(tensor_output.host_view());

    tensor_weight.sync_device();
    tensor_activations.sync_device();
    tensor_rows.sync_device();
    tensor_bias.sync_device();
    tensor_output.sync_device();
  }

  /// Verifies the result is a GEMV
  bool verify_(double& diff) {

    bool passed = true;

    tensor_output.sync_host();

    // Reference check
    // passed = cutlass::reference::host::TensorEquals(tensor_output.host_view(), reference_output.host_view());
    diff = cutlass::reference::host::TensorSumSqDiff(tensor_output.host_view(), reference_output.host_view());
    passed = diff / options.batch / options.N < 1e-2;

    if (!passed) {
      std::cerr << "\n***\nError - problem failed the QA check\n***\n" << std::endl;

      std::stringstream fname;

      fname << std::filesystem::path(__FILE__).parent_path().string()
            << "/error_97_pit_gemv_"
            << options.batch << "x"
            << options.N << "x"
            << options.K << ".txt";

      std::cout << fname.str() << std::endl;

      std::ofstream results(fname.str());

      results
        // << "\nWeight:\n" << tensor_weight.host_view() << "\n"
        << "\nActivations:\n" << tensor_activations.host_view() << "\n"
        << "\nOutput Reference:\n" << reference_output.host_view() << "\n"
        << "\nOutput Computed:\n" << tensor_output.host_view() << "\n";
    }

    return passed;
  }

public:

  /// Returns the number of threadblocks to launch if the kernel can run on the target
  /// device. Otherwise, returns zero.
  bool sufficient() const {
    //
    // Determine SMEM requirements and waive if not satisfied
    //

    int smem_size = int(sizeof(typename Gemv::GemvKernel::SharedStorage));

    hipDeviceProp_t properties;
    int device_idx;
    hipError_t result = hipGetDevice(&device_idx);

    if (result != hipSuccess) {
      throw std::runtime_error("hipGetDevice() API call failed.");
    }

    result = hipGetDeviceProperties(&properties, device_idx);

    if (result != hipSuccess) {
      throw std::runtime_error("hipGetDeviceProperties() failed");
    }

    if (properties.sharedMemPerBlockOptin < smem_size) {
      return false;
    }

    return true;
  }

  /// Executes a PIT GeMV kernel and measures runtime.
  Result profile() {

    Result result;

    // Early exit
    if (!sufficient()) {
      std::cout << "Active CUDA device lacks hardware resources to run PIT GeMV kernel." << std::endl;
      return result;
    }

    result.passed = false;

    // Initialize the problem
    initialize_();

    MatrixCoord problem_size = {options.K, options.N};
    std::cout << "Problem Size: Row=" << problem_size.row() << ", Column=" << problem_size.column() << std::endl;
    // Configure GEMV arguments
    typename Gemv::Arguments args(
      problem_size,
      options.batch,
      tensor_weight.device_ref(),
      tensor_activations.device_data(),
      tensor_bias.device_data(),
      tensor_output.device_data(),
      options.rows,            // rows_count
      tensor_rows.device_data(),
      0,                       // batch_stride_A
      options.K,               // batch_stride_B
      options.N,               // batch_stride_C
      options.N                // batch_stride_D
    );

    // Initialize the GEMV object
    Gemv gemv;

    result.status = gemv.initialize(args);

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize PIT GeMV kernel." << std::endl;
      return result;
    }

    // Run the PIT GeMV object
    result.status = gemv.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run PIT GeMV kernel." << std::endl;
      return result;
    }

    // Wait for completion
    result.error = hipDeviceSynchronize();

    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    //
    // Verify correctness
    //
    result.passed = verify_(result.diff);

    result.error = hipDeviceSynchronize();
    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    //
    // Warm-up run
    //
    result.status = gemv.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run PIT GeMV kernel." << std::endl;
      return result;
    }

    result.error = hipDeviceSynchronize();
    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    // Record an event at the start of a series of GEMV operations
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Run profiling loop
    //

    result.error = hipDeviceSynchronize();
    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    for (int iter = 0; iter < options.iterations; ++iter) {
      gemv();
    }

    result.error = hipDeviceSynchronize();
    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMV operations have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Compute average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    //
    // Cleanup
    //

    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    std::cout << std::endl;
    std::cout << "PIT GeMV (CUTLASS):\n"
      << "       NxK = " << options.N << "x" << options.K << "(" << options.rows << ")\n"
      << "====================================================" << std::endl;

    std::cout << std::endl;
    std::cout << "    " << "SumDiff: " << result.diff << std::endl;
    std::cout << "    " << "Runtime: " << result.runtime_ms << " ms" << std::endl;
    std::cout << "    " << " GFLOPs: " << result.gflops << std::endl;

    return result;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 11 || props.major < 8) {
  
    //
    // This example requires an NVIDIA Ampere-architecture GPU.
    //

    std::cout 
      << "PIT GeMV example requires a GPU of NVIDIA's Ampere Architecture or "
      << "later (compute capability 80 or greater).\n";

    return 0;
  }

  //
  // Parse options
  //

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  using ElementA = cutlass::half_t;
  using ElementB = cutlass::half_t;
  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using LayoutA = cutlass::layout::RowMajor;

  int const kElementsPerAccess = 16;
  int const kThreadCount = 256;
  int const kThreadsPerCol = 64;

  using GemvKernel = cutlass::gemm::kernel::PitGemv<ElementA,
                                                    LayoutA,
                                                    ElementB,
                                                    ElementOutput,
                                                    ElementAccumulator,
                                                    kElementsPerAccess,
                                                    kThreadCount,
                                                    kThreadsPerCol>;
  using Gemv = cutlass::gemm::device::PitGemv<GemvKernel>;

  //
  // Profile it
  //

  Testbed<Gemv> testbed(options);

  if (!testbed.sufficient()) {
    std::cout << "The active CUDA device lacks sufficient hardware resources to execute this kernel.\n";
    return 0;
  }

  Result result = testbed.profile();
  if (!result.passed) {
    std::cout << "\nFailed tp profile PIT GEMV.\n";
    return -1;
  }

  std::cout << "\nPassed\n";

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
